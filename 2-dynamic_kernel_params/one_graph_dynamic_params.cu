#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "sgemm_utils.h"
#include <chrono>
#include <iostream>
void random_matrix(int m, int n, int k, float *A, float *B)
{
    int lda = k;
    int ldb = n;
    // A matrix init
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < k; j++)
        {
            A(i, j) = 2.0 * drand48() - 1.0;
            // A(i, j) = 1.0;
        }
    }
    // B matrix init
    for (int i = 0; i < k; i++)
    {
        for (int j = 0; j < n; j++)
        {
            B(i, j) = 2.0 * drand48() - 1.0;
            // B(i, j) = 1.0;
        }
    }
}

__device__ inline float warp_reduce_sum(float val)
{
    for (int offset = 32 / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);

    return val;
}

__global__ void sgemm(int M, int N, int K, float *A, float *B, float *C)
{

    int offset = threadIdx.x;
    const int x = blockIdx.x;
    const int y = blockIdx.y;

    float *A_ptr_start = A + blockIdx.y * K;
    float *B_ptr_start = B + blockIdx.x;
    float temp = 0.f;
    for (int k = offset; k < K; k += blockDim.x)
    {
        temp += A_ptr_start[k] * B_ptr_start[k * N]; // 注意 k 是A的列，B的行
    }

    C[x + y * N] = warp_reduce_sum(temp);
}

int main()
{
    int M = 512;
    int N = 512;
    int K = 512;

    const size_t mem_size_A = M * K * sizeof(float);
    const size_t mem_size_B = K * N * sizeof(float);
    const size_t mem_size_C = M * N * sizeof(float);

    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);

    float *matrix_C_gpu_host = (float *)malloc(mem_size_C);
    float *matrix_C_cpu_host = (float *)malloc(mem_size_C);

    random_matrix(M, N, K, matrix_A_host, matrix_B_host);

    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    // 获取释放前的内存信息
    size_t freeMemBefore, totalMem;
    hipMemGetInfo(&freeMemBefore, &totalMem);
    printf("Before freeing: Free %lu, Total: %lu\n", freeMemBefore, totalMem);

    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);

    hipGraph_t cgraph;
    hipStream_t cstream;
    hipStreamCreate(&cstream);
    hipGraphExec_t instance;

    // 开始在这个流里捕获图
    hipStreamBeginCapture(cstream, hipStreamCaptureModeGlobal);

    // func
    {
        hipKernelNodeParams params;
        params.blockDim = {32, 1, 1};
        params.gridDim = {static_cast<unsigned>(M), static_cast<unsigned int>(N), 1};
        params.sharedMemBytes = 0;
        params.extra = nullptr;
        params.func = reinterpret_cast<void *>(sgemm);

        void *kenelParams[] = {
            &M,
            &N,
            &K,
            &matrix_A_device,
            &matrix_B_device,
            &matrix_C_device};

        params.kernelParams = kenelParams;
        hipStreamCaptureStatus capture_status;
        const hipGraphNode_t *deps;
        size_t dep_count;
        cudaStreamGetCaptureInfo_v2(cstream, &capture_status, nullptr, &cgraph, &deps, &dep_count);

        hipGraphNode_t new_node;
        hipGraphAddKernelNode(&new_node, cgraph, deps, dep_count, &params);
    }

    hipStreamEndCapture(cstream, &cgraph);
    hipGraphInstantiate(&instance, cgraph, nullptr, nullptr, 0);

    hipGraphLaunch(instance, cstream);

    hipMemcpy(matrix_C_gpu_host, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);

    printf("%f\n", matrix_C_gpu_host[0]);

    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_cpu_host);
    free(matrix_C_gpu_host);
    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
    std::cout << "free device buffer" << std::endl;
    // 获取释放后的内存信息
    size_t freeMemAfter;
    hipMemGetInfo(&freeMemAfter, &totalMem);
    printf("After freeing: Free %lu, Total: %lu\n", freeMemAfter, totalMem);

    return 0;
}